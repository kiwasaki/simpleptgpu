#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<memory>
#include<chrono>

#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include"ray.hpp"
#include"sphere.hpp"
#include"scene.hpp"
#include"image.hpp"

constexpr float pi = 3.14159265f;
constexpr float two_pi = 2.f * pi;
constexpr float inv_pi = 0.318309886f;

//
__global__ void make_scene( sphere **device_spheres, scene **device_scene, const int n )
{
	if( threadIdx.x == 0 && blockIdx.x == 0 ) {
		int k = 0;

		//device_spheres[ 0 ] = new sphere( make_float3( 0.f, 1.0f, 10.f ), 1.f, make_float4( 1.f, 1.f, 1.f, 0.1f ) );
		device_spheres[ k++ ] = new sphere( make_float3( 0.f, -1e3f, 0.f ), 1e3f, make_float4( 1.f, 1.f, 1.f, 0.1f ) ); //floor
		device_spheres[ k++ ] = new sphere( make_float3( 1e3f + 5.f, 0.f, 0.f ), 1e3f, make_float4( 0.8f, 0.2f, 0.1f, 0.1f ) ); //right
		device_spheres[ k++ ] = new sphere( make_float3( - 1e3f - 5.f, 0.f, 0.f ), 1e3f, make_float4( 0.1f, 0.2f, 0.8f, 0.1f ) ); //left
		device_spheres[ k++ ] = new sphere( make_float3( 0.f, 0.f, 1e3f + 15.f ), 1e3f, make_float4( 1.f, 1.f, 1.f, 0.1f ) ); //far
		device_spheres[ k++ ] = new sphere( make_float3( 0.f, 1e3f + 8, 0.f ), 1e3f, make_float4( 1.f, 1.f, 1.f, 0.01f ) ); //ceil
		device_spheres[ k++ ] = new sphere( make_float3( 0.f, 7.0f, 10.f ), 1.f, make_float4( 10.f, 10.f, 10.f, - 1.f ) ); //light source
		device_spheres[ k++ ] = new sphere( make_float3( 0.f, 1.0f, 10.f ), 1.f, make_float4( 1.f, 1.f, 1.f, 0.9f ) );

		*device_scene = new scene( device_spheres, k );
	}
}


//
__global__ void free_scene( sphere **device_spheres, scene **device_scene, const int n )
{
	if( threadIdx.x == 0 && blockIdx.x == 0 ) {
		for( int i = 0; i < n; ++i ) {
			delete device_spheres[ i ];
		}
		delete *device_scene;
	}
}

//
__global__ void trace( float *pixels, scene **scene, const int width, const int height, const float3 eye )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( ( x >= width ) && ( y >= height ) ) return;

	const float m_p = 2.f * tan( 40.f / 2.f * 3.14159265f / 180.f ) / float( height );
	const float3 d = { m_p * ( x - width / 2.f ), m_p * ( y - height / 2.f ), 1.f };
	const ray r = { eye, normalize( d ) };

	intersection isect;

	bool hit = ( *scene )->intersect( r, isect );
	if( hit ) {
		pixels[ 3 * ( y * width + x ) + 0 ] = 0.5f * ( isect.m_n.x + 1.f );
		pixels[ 3 * ( y * width + x ) + 1 ] = 0.5f * ( isect.m_n.y + 1.f );
		pixels[ 3 * ( y * width + x ) + 2 ] = 0.5f * ( isect.m_c.z + 1.f );
	} else {
		pixels[ 3 * ( y * width + x ) + 0 ] = 0.f;
		pixels[ 3 * ( y * width + x ) + 1 ] = 0.f;
		pixels[ 3 * ( y * width + x ) + 2 ] = 0.f;
	}
}

//
__global__ void render_aa( float *pixels, scene **scene, hiprandState *rand_state, const int width, const int height, const float3 eye, const int ns )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( ( x >= width ) && ( y >= height ) ) return;

	intersection isect;
	float3 d, L;
	ray r;
	hiprandState rng = rand_state[ y * width + x ];

	const float m_p = 2.f * tan( 40.f / 2.f * 3.14159265f / 180.f ) / float( height );

	L = make_float3( 0.f, 0.f, 0.f );
	for( int i = 0; i < ns; ++i ) {
		d = { m_p * ( float( x ) - float( width ) / 2.f + hiprand_uniform( &rng ) ), m_p * ( float( y ) - float( height ) / 2.f + hiprand_uniform( &rng ) ), 1.f };
		//d = { m_p * ( x - width / 2.f + 0.5f ), m_p * ( y - height / 2.f + 0.5f ), 1.f };
		r = { eye, normalize( d ) };
		if( ( *scene )->intersect( r, isect ) ) {
			L.x += 0.5f * ( isect.m_n.x + 1.f );
			L.y += 0.5f * ( isect.m_n.y + 1.f );
			L.z += 0.5f * ( isect.m_n.z + 1.f );
		}
	}
	pixels[ 3 * ( y * width + x ) + 0 ] = L.x / float( ns );
	pixels[ 3 * ( y * width + x ) + 1 ] = L.y / float( ns );
	pixels[ 3 * ( y * width + x ) + 2 ] = L.z / float( ns );
}


//
__global__ void init( hiprandState *rand_state, const int width, const int height )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( ( x >= width ) || ( y >= height ) ) return;

	//hiprand_init( 1984, y * width + x, 0, &rand_state[ y * width + x ] );
	hiprand_init( 1984 + y * width + x, 0, 0, &rand_state[ y * width + x ] );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv )
{
	constexpr int width  = 1024;
	constexpr int height = 1024;

	constexpr int n_object	= 7;

	float *device_buffer;

	sphere **device_spheres;
	scene  **device_scene;
	hiprandState *device_rand_state;

	//
	const float3 eye = { 0.f, 3.f, - 5.f };

	//
	checkCudaErrors( hipMallocManaged( ( void ** ) &device_buffer, sizeof( float ) * 3 * width * height ) );

	//
	checkCudaErrors( hipMalloc( ( void** ) &device_rand_state, sizeof( hiprandState ) * width * height ) );

	//シーンの作成
	{
		checkCudaErrors( hipMalloc( ( void ** ) &device_spheres, n_object * sizeof( sphere * ) ) );
		checkCudaErrors( hipMalloc( ( void ** ) &device_scene  , 1 * sizeof( scene *  ) ) );
		make_scene<<< 1, 1 >>>( device_spheres, device_scene, n_object );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	const auto start = std::chrono::system_clock::now();

	{
		dim3 grid, block;
		block.x = 16;
		block.y = 16;
		grid.x = width  / block.x;
		grid.y = height / block.y;
		checkCudaErrors( hipMalloc( ( void ** ) &device_rand_state, width * height * sizeof( hiprandState ) ) );
		init<<< grid, block >>>( device_rand_state, width, height );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	{
		dim3 grid, block;
		block.x = 32;
		block.y = 32;
		grid.x = width  / block.x;
		grid.y = height / block.y;

		render_aa<<< grid, block >>>( device_buffer, device_scene, device_rand_state, width, height, eye, 10 );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}
	const auto end = std::chrono::system_clock::now();
	std::cout << std::chrono::duration_cast< std::chrono::milliseconds >( end - start ).count() << "ms.\n";

	{
		free_scene<<< 1, 1 >>>( device_spheres, device_scene, n_object );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	//
	save_bmp( device_buffer, width, height, "rtaa.bmp" );

	checkCudaErrors( hipFree( device_rand_state ) );
	checkCudaErrors( hipFree( device_spheres ) );
	checkCudaErrors( hipFree( device_scene ) );
	checkCudaErrors( hipFree( device_buffer ) );

	return 0;

}
