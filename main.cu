#include "hip/hip_runtime.h"
#include<iostream>
#include<vector>
#include<memory>

#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include"ray.hpp"
#include"sphere.hpp"
#include"scene.hpp"
#include"image.hpp"

//
__global__ void make_scene( sphere **device_spheres, scene **device_scene, const int n )
{
	if( threadIdx.x == 0 && blockIdx.x == 0 ) {
		device_spheres[ 0 ] = new sphere( make_float3( 0.f, 1.0f, 10.f ), 1.f, make_float4( 1.f, 1.f, 1.f, 1.f ) );
		device_spheres[ 1 ] = new sphere( make_float3( 0.f, -1e4f, 0.f ), 1e4f, make_float4( 1.f, 1.f, 0.f, 1.f ) );
		device_spheres[ 2 ] = new sphere( make_float3( 0.f, 7.f, 10.f ), 1.f, make_float4( 10.f, 0.f, 0.f, 0.f ) );
		*device_scene = new scene( device_spheres, n );
	}
}

//
__global__ void free_scene( sphere **device_spheres, scene **device_scene, const int n )
{
	if( threadIdx.x == 0 && blockIdx.x == 0 ) {
		for( int i = 0; i < n; ++i ) {
			delete device_spheres[ i ];
		}
		delete *device_scene;
	}
}

//
__global__ void trace( float *pixels, scene **scene, const int width, const int height, const float3 eye )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( ( x >= width ) && ( y >= height ) ) return;

	const float m_p = 2.f * tan( 40.f / 2.f * 3.14159265f / 180.f ) / float( height );
	const float3 d = { m_p * ( x - width / 2.f ), m_p * ( y - height / 2.f ), 1.f };
	const ray r = { eye, normalize( d ) };

	intersection isect;

	bool hit = ( *scene )->intersect( r, isect );
	if( hit ) {
		pixels[ 3 * ( y * width + x ) + 0 ] = 0.5f * ( isect.m_n.x + 1.f );
		pixels[ 3 * ( y * width + x ) + 1 ] = 0.5f * ( isect.m_n.y + 1.f );
		pixels[ 3 * ( y * width + x ) + 2 ] = 0.5f * ( isect.m_c.z + 1.f );
	} else {
		pixels[ 3 * ( y * width + x ) + 0 ] = 0.f;
		pixels[ 3 * ( y * width + x ) + 1 ] = 0.f;
		pixels[ 3 * ( y * width + x ) + 2 ] = 0.f;
	}
}



//
__global__ void render_aa( float *pixels, scene **scene, hiprandState *rand_state, const int width, const int height, const float3 eye, const int ns )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( ( x >= width ) && ( y >= height ) ) return;

	intersection isect;
	float3 d, L;
	ray r;
	bool hit = false;

	const float m_p = 2.f * tan( 40.f / 2.f * 3.14159265f / 180.f ) / float( height );

	L = make_float3( 0.f, 0.f, 0.f );
	for( int i = 0; i < ns; ++i ) {
		d = { m_p * ( x - width / 2.f + hiprand_uniform( &rand_state[ y * width + x ] ) ), m_p * ( y - height / 2.f + hiprand_uniform( &rand_state[ y * width + x ] ) ), 1.f };
		r = { eye, normalize( d ) };
		hit = ( *scene )->intersect( r, isect );
		if( hit ) {
			L.x += 0.5f * ( isect.m_n.x + 1.f );
			L.y += 0.5f * ( isect.m_n.y + 1.f );
			L.z += 0.5f * ( isect.m_n.z + 1.f );
		}
	}
	pixels[ 3 * ( y * width + x ) + 0 ] = L.x / float( ns );
	pixels[ 3 * ( y * width + x ) + 1 ] = L.y / float( ns );
	pixels[ 3 * ( y * width + x ) + 2 ] = L.z / float( ns );
}



//
__global__ void init( hiprandState *rand_state, const int width, const int height )
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	if( ( x >= width ) || ( y >= height ) ) return;

	hiprand_init( 1984, y * width + x, 0, &rand_state[ y * width + x ] );
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv )
{
	constexpr int width		= 512;
	constexpr int height	= 512;

	constexpr int n_object	= 3;

	float 						*device_image;
	std::unique_ptr< float [] > host_image;
	sphere **device_spheres;
	scene  **device_scene;
	hiprandState *device_rand_state;

	//
	const float3 eye = { 0.f, 3.f, - 5.f };

	//
	checkCudaErrors( hipMalloc( ( void** ) &device_rand_state, sizeof( hiprandState ) * width * height ) );

	//
	host_image = std::make_unique< float [] >( 3 * width * height );
	checkCudaErrors( hipMalloc( ( void ** ) &device_image, sizeof( float ) * 3 * width * height ) );

	//シーンの作成
	{
		checkCudaErrors( hipMalloc( ( void ** ) &device_spheres, n_object * sizeof( sphere * ) ) );
		checkCudaErrors( hipMalloc( ( void ** ) &device_scene  , 1 * sizeof( scene *  ) ) );
		make_scene<<< 1, 1 >>>( device_spheres, device_scene, n_object );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	{
		dim3 grid, block;
		block.x = 16;
		block.y = 16;
		grid.x = width  / block.x;
		grid.y = height / block.y;
		checkCudaErrors( hipMalloc( ( void ** ) &device_rand_state, width * height * sizeof( hiprandState ) ) );
		init<<< grid, block >>>( device_rand_state, width, height );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}


	{
		dim3 grid, block;
		block.x = 16;
		block.y = 16;
		grid.x = width  / block.x;
		grid.y = height / block.y;

		//trace<<< grid, block >>>( device_image, device_scene, width, height, eye );
		render_aa<<< grid, block >>>( device_image, device_scene, device_rand_state, width, height, eye, 10 );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	{
		free_scene<<< 1, 1 >>>( device_spheres, device_scene, n_object );
		checkCudaErrors( hipGetLastError() );
		checkCudaErrors( hipDeviceSynchronize() );
	}

	checkCudaErrors( hipMemcpy( host_image.get(), device_image, sizeof( float ) * 3 * width * height, hipMemcpyDeviceToHost ) );

	save_bmp( host_image.get(), width, height, "test.bmp" );

	//delete
	checkCudaErrors( hipFree( device_image ) );
	checkCudaErrors( hipFree( device_rand_state ) );
	checkCudaErrors( hipFree( device_spheres ) );
	checkCudaErrors( hipFree( device_scene ) );

	return 0;

}
